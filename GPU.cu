#include "hip/hip_runtime.h"
#include <iostream>
#include <string>

/* Cuda stuff */
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

#include "GPU.h"
#include "TrainingSet.h"
#include "StrucClassSSF.h"

using namespace vision;

void check_cuda( hipError_t ok, std::string message )
{
	if( ok!=hipSuccess )
		std::cerr << ">>> Error: " << message << std::endl;
}

__device__
float gpuGetValueIntegral (float *gpuFeaturesIntegral, uint8_t channel, 
	int16_t x1, int16_t y1, int16_t x2, int16_t y2, int16_t w, int16_t h)
{
	float res = (
			gpuFeaturesIntegral[y2 + x2*h + channel*w*h] -
			gpuFeaturesIntegral[y2 + x1*h + channel*w*h] -
			gpuFeaturesIntegral[y1 + x2*h + channel*w*h] +
			gpuFeaturesIntegral[y1 + x1*h + channel*w*h]);

	return res;
}

/***************************************************************************
 Prepare the kernel call:
 - Transfer the features to the GPU
 - Prepare an array for the results, initialized to zero (in parallel on the GPU)
 ***************************************************************************/
void preKernel(float *features, float *features_integral,
	float **_gpuFeatures, float **_gpuFeaturesIntegral, unsigned int **_gpuResult,
	int16_t w, int16_t h, int16_t w_integral, int16_t h_integral, int16_t noChannels, 
	int numLabels, int16_t numTries, StrucClassSSF<float> *forest, StrucClassSSF<float> **_gpuForest)
{
	hipError_t ok;
	int size;

	// Init GPU memory for the features
	size = noChannels*w*h*sizeof(float);
	ok = hipMalloc ((void**) _gpuFeatures, size);
	check_cuda(ok, err_alloc);

	ok = hipMemcpy (*_gpuFeatures, features, size, hipMemcpyHostToDevice);
	check_cuda(ok, err_cpy);

	size = noChannels*w_integral*h_integral*sizeof(float);
	ok = hipMalloc ((void**) _gpuFeaturesIntegral, size);
	check_cuda(ok, err_alloc);

	ok = hipMemcpy (*_gpuFeaturesIntegral, features_integral, size, hipMemcpyHostToDevice);
	check_cuda(ok, err_cpy);
	
	// Forest load
	size = 0;
	for( int i=0; i < numTries; i++ )
		size += sizeof(forest[i]) - sizeof(int*) + forest[i].getHeapSize();
	//ok = hipMalloc ((void**) _gpuForest, size);
	//check_cuda(ok, err_alloc);
	// TODO copy heap
	//ok = hipMemcpy (*_gpuForest, forest, size, hipMemcpyHostToDevice);
	//check_cuda(ok, err_cpy);

	// Allocate memory for the results
	size=w*h*numLabels*sizeof(unsigned int);
	ok=hipMalloc ((void**) _gpuResult, size);
	check_cuda(ok, err_alloc);

}


/***************************************************************************
 After the kernel call:
 - Transfer the result back from the GPU to the _CPU
 - free the GPU memory related to a single image
 ***************************************************************************/
void postKernel(float *_gpuFeatures, float *_gpuFeaturesIntegral, unsigned int *_gpuResult,
	unsigned int *result, int16_t w, int16_t h, int numLabels)
{
	hipError_t ok;
	int size;

	// Copy the results back to host memory
	size=w*h*numLabels*sizeof(unsigned int);
	ok=hipMemcpy (result, _gpuResult, size, hipMemcpyDeviceToHost);
	check_cuda(ok, err_cpy);

#ifdef GPU_DEBUG_SINGLE_PIXEL
	std::cerr << "Debug-error code (int)=" << std::dec << (int) *result << "\n";
	std::cerr << "Return values: ";
	for (int i=0; i<result[0]; ++i)
		std::cerr << result[i+1] << " ";
	std::cerr << "\n";
#endif  		

	// Free GPU memory.
	hipFree(_gpuFeatures);
	hipFree(_gpuFeaturesIntegral);
	hipFree(_gpuResult);
	// TODO free the forest
}
